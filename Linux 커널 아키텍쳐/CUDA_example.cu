
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// 해당예제는 엔비디아 CUDA를 사용한 간단한 예시로 두 벡터의 각 요소를 더하는 기본적인 병렬 덧셈 프로그램


// CUDA 커널 함수 정의
__global__ void add(int n, float *x, float *y) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<20; // 1M 요소
    float *x, *y, *d_x, *d_y;

    // 호스트 메모리 할당 -> CPU에 메모리를 할당한다. 
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    // GPU 메모리 할당 -> 필요한 공간을 할당하고 초기화 시킨다. 
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    // 호스트에서 데이터 초기화
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // 데이터를 GPU로 복사 -> 초기화된 데이터를 호스트(CPU)에서 디바이스로 이동시킨다. 
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // CUDA 커널 실행
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, d_x, d_y);

    // 결과를 호스트로 복사
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // 결과 확인
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // 메모리 해제
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    
    return 0;
}
