
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

// 해당예제는 엔비디아 CUDA를 사용한 간단한 예시로 두 벡터의 각 요소를 더하는 기본적인 병렬 덧셈 프로그램


// CUDA 커널 함수 정의
__global__ void add(int n, float *x, float *y) {            // __global__ 키워드는 이 함수가 GPU에서 실행될 CUDA 커널임을 나타낸다. 
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

int main(void) {
    int N = 1<<20; // 1M 요소
    float *x, *y, *d_x, *d_y;

    // 호스트 메모리 할당 -> CPU에 메모리를 할당한다. 
    x = (float*)malloc(N*sizeof(float));
    y = (float*)malloc(N*sizeof(float));

    // GPU 메모리 할당 -> 필요한 공간을 할당하고 초기화 시킨다. d_x, d_y 는 GPU 메모리 내의 배열을 가리키는 포인터 
    hipMalloc(&d_x, N*sizeof(float)); 
    hipMalloc(&d_y, N*sizeof(float));

    // 호스트에서 데이터 초기화
    for (int i = 0; i < N; i++) {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    // 데이터를 GPU로 복사 -> 초기화된 데이터를 호스트(CPU)에서 디바이스(GPU)로 이동시킨다. 
    hipMemcpy(d_x, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, N*sizeof(float), hipMemcpyHostToDevice);

    // CUDA 커널 실행
    int blockSize = 256;
    int numBlocks = (N + blockSize - 1) / blockSize;
    add<<<numBlocks, blockSize>>>(N, d_x, d_y);

    // 결과를 호스트로 복사
    hipMemcpy(y, d_y, N*sizeof(float), hipMemcpyDeviceToHost);

    // 결과 확인
    float maxError = 0.0f;
    for (int i = 0; i < N; i++)
        maxError = fmax(maxError, fabs(y[i]-3.0f));
    std::cout << "Max error: " << maxError << std::endl;

    // 메모리 해제
    hipFree(d_x);
    hipFree(d_y);
    free(x);
    free(y);
    
    return 0;
}



// CUDA는 딥 러닝 분야에서 주로 대규모 신경망 모델의 훈련과 추론에 사용됩니다. 딥 러닝 모델, 특히 깊은 신경망과 컨볼루션 신경망은 많은 양의 데이터와 복잡한 계산이 필요하기 때문에, CUDA를 사용한 GPU 가속은 이러한 모델을 훈련하고 실행하는 데 있어서 중요한 역할을 합니다.
// torch 와 연계해서도 사용할 수 있는것 같음 